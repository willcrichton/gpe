#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

typedef struct {
  float x;
  float y;
} Point;

typedef struct {
  Point points[];
  int num_points;
} Polygon;

typedef struct {
  Polygon polygons[];
  int num_polygons;
} Encoding;

__global__ void render_kernel() {

}

extern "C" void cuda_render(Encoding img) {
  printf("%d\n", img.num_polygons);
  //printf("%f, %f", p.x, p.y);
}