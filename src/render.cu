#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREADS_PER_BLOCK 128

typedef unsigned int u32;
typedef unsigned char u8;

typedef struct {
  u8 r;
  u8 g;
  u8 b;
} Color;

typedef struct {
  float x;
  float y;
} Point;

typedef struct {
  Point pos;
  Color color;
} Pixel;

typedef struct {
  Point* points;
  u32 num_points;
  u8 r; u8 g; u8 b; u8 a;
  Point center;
  float max_dist;
} Polygon;

typedef struct {
  Polygon* polygons;
  u32 num_polygons;
  Pixel* pixels;
  u32 num_pixels;
  u32 width;
  u32 height;
  Polygon* dev_poly_ptr;
  Pixel* dev_pixel_ptr;
} Encoding;

typedef struct {
  bool contains;
  float distance;
} QueryResult;

__device__ __inline__ u8 add(u32 older, u32 newer, u32 alpha) {
  u32 addend = newer * alpha / 255;
  if (addend + older > 255) { /*printf("%d, %d, %d\n", older, newer, alpha);*/ return 255; }
  else { return addend + older; }
}

__device__ __inline__ void blend(Color* old_color, Color* new_color, u32 alpha) {
  old_color->r = add(old_color->r, new_color->r, alpha);
  old_color->g = add(old_color->g, new_color->g, alpha);
  old_color->b = add(old_color->b, new_color->b, alpha);
}

__device__ __inline__ Color polycolor(Polygon poly, Point pt) {
  Color color;
  float x = pt.x - poly.center.x;
  float y = pt.y - poly.center.y;
  float scale = 1.0 - (x * x + y * y) / poly.max_dist;

  color.r = poly.r * scale;
  color.g = poly.g * scale;
  color.b = poly.b * scale;
  return color;
}

__device__ __inline__ float distance_squared(Point a, Point b) {
  float dx = a.x - b.x, dy = a.y - b.y;
  return dx * dx + dy * dy;
}

__device__ QueryResult query(Point pt, Polygon poly, bool antialias) {
  QueryResult result;
  result.contains = false;
  result.distance = 1000000.0;
  for (int i = 0; i < poly.num_points; i++) {
    Point a = poly.points[i], b = poly.points[(i + 1) % poly.num_points];
    if ((a.y > pt.y) != (b.y > pt.y) &&
        (pt.x < (b.x - a.x) * (pt.y - a.y) / (b.y - a.y) + a.x)) {
      result.contains = !result.contains;
    }

    if (antialias) {
      float mag = distance_squared(a, b);
      float t = ((pt.x - a.x) * (b.x - a.x) + (pt.y - a.y) * (b.y - a.y)) / mag;
      float dist;
      if (t < 0.0) {
        dist = distance_squared(pt, a);
      } else if (t > 1.0) {
        dist = distance_squared(pt, b);
      } else {
        Point ba;
        ba.x = a.x + (b.x - a.x) * t;
        ba.y = a.y + (b.y - a.y) * t;
        dist = distance_squared(pt, ba);
      }

      if (dist < result.distance) {
        result.distance = dist;
      }
    }
  }

  return result;
}

__global__ void render_kernel(Encoding* img, Color* output, bool antialias) {
  int pixel = blockDim.x * blockIdx.x + threadIdx.x;
  if (pixel >= img->width * img->height) return;

  Point pt = { pixel % img->width, pixel / img->width };

  for (int i = 0; i < img->num_polygons; i++) {
    Polygon polygon = img->polygons[i];

    QueryResult result = query(pt, polygon, antialias);
    if (result.contains || (antialias && result.distance < 4.0)) {
      float alpha = polygon.a;
      if (!result.contains) {
        alpha /= ((1.0 + result.distance) * (1.0 + result.distance));
      }

      blend(&output[pixel], &polycolor(polygon, pt), alpha);
    }
  }

  for (int i = 0; i < img->num_pixels; i++) {
    Pixel p = img->pixels[i];
    if (p.pos.x == pt.x && p.pos.y == pt.y) {
      output[pixel].r = p.color.r;
      output[pixel].g = p.color.g;
      output[pixel].b = p.color.b;
    }
  }
}

Point* points_to_cuda(Polygon polygon) {
  Point* cuda_points;
  hipMalloc(&cuda_points, sizeof(Point) * polygon.num_points);
  hipMemcpy(cuda_points, polygon.points, sizeof(Point) * polygon.num_points, hipMemcpyHostToDevice);

  return cuda_points;
}

Polygon* polygons_to_cuda(Encoding* img) {
  Polygon* cuda_polygons;
  hipMalloc(&cuda_polygons, sizeof(Polygon) * img->num_polygons);

  for (int i = 0; i < img->num_polygons; i++) {
    float max_dist = 0.0;
    for (int j = 0; j < img->polygons[i].num_points; j++) {
      float x = img->polygons[i].points[j].x - img->polygons[i].center.x;
      float y = img->polygons[i].points[j].y - img->polygons[i].center.y;
      float dist = x * x + y * y;
      if (dist > max_dist) {
        max_dist = dist;
      }
    }

    img->polygons[i].max_dist = max_dist;
    img->polygons[i].points = points_to_cuda(img->polygons[i]);
  }

  hipMemcpy(cuda_polygons, img->polygons, sizeof(Polygon) * img->num_polygons, hipMemcpyHostToDevice);

  return cuda_polygons;
}

Pixel* pixels_to_cuda(Encoding *img) {
  Pixel* cuda_pixels;
  hipMalloc(&cuda_pixels, sizeof(Pixel) * img->num_pixels);
  hipMemcpy(cuda_pixels, img->pixels, sizeof(Pixel) * img->num_pixels, hipMemcpyHostToDevice);

  return cuda_pixels;
}

Encoding* encoding_to_cuda(Encoding* img) {
  Encoding* cuda_img;
  hipMalloc(&cuda_img, sizeof(Encoding));

  Polygon* poly_tmp = img->polygons;
  img->polygons = polygons_to_cuda(img);

  Pixel* pixel_tmp = img->pixels;
  img->pixels = pixels_to_cuda(img);

  hipMemcpy(cuda_img, img, sizeof(Encoding), hipMemcpyHostToDevice);

  img->dev_pixel_ptr = img->pixels;
  img->dev_poly_ptr = img->polygons;
  img->polygons = poly_tmp;
  img->pixels = pixel_tmp;
  return cuda_img;
}

void encoding_free(Encoding* img) {
  for (int i = 0; i < img->num_polygons; i++) {
    hipFree(img->polygons[i].points);
  }

  hipFree(img->dev_pixel_ptr);
  hipFree(img->dev_poly_ptr);
}

extern "C" void cuda_render(Encoding img, Color* output, bool antialias) {
  u32 N = img.width * img.height;
  size_t size = N * sizeof(Color);

  Color* cuda_output;
  hipMalloc(&cuda_output, size);
  hipMemcpy(cuda_output, output, size, hipMemcpyHostToDevice);

  Encoding* cuda_img = encoding_to_cuda(&img);

  dim3 threadsPerBlock(THREADS_PER_BLOCK, 1);
  dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x);
  render_kernel<<<blocksPerGrid, threadsPerBlock>>>(cuda_img, cuda_output, antialias);

  hipMemcpy(output, cuda_output, size, hipMemcpyDeviceToHost);

  encoding_free(&img);
  hipFree(cuda_img);
  hipFree(cuda_output);
}