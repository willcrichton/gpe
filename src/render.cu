#include <hip/hip_runtime.h>
#include <stdio.h>

typedef unsigned int uint;

typedef struct {
  char r;
  char g;
  char b;
} Color;

typedef struct {
  float x;
  float y;
} Point;

typedef struct {
  Point* points;
  uint num_points;
  char r; char g; char b; char a;
} Polygon;

typedef struct {
  Polygon* polygons;
  uint num_polygons;
  uint width;
  uint height;
} Encoding;

__device__ __inline__ char add(uint older, uint newer, uint alpha) {
  uint addend = newer * alpha / 255;
  if (addend + older > 255) { return 255; }
  else { return addend + older; }
}

__device__ __inline__ Color blend(Color old_color, Polygon poly) {
  Color color;
  color.r = add(old_color.r, poly.r, poly.a);
  color.g = add(old_color.g, poly.g, poly.a);
  color.b = add(old_color.b, poly.b, poly.a);
  return color;
}

__global__ void render_kernel(Encoding* img, Color* output, bool antialias) {
  int pixel = blockDim.x * blockIdx.x + threadIdx.x;
  if (pixel >= img->width * img->height) return;

  for (int i = 0; i < img->num_polygons; i++) {
    Polygon polygon = img->polygons[i];

    // TODO: polygon containment test
    output[pixel] = blend(output[pixel], polygon);
  }
}

Point* points_to_cuda(Polygon polygon) {
  Point* cuda_points;
  hipMalloc(&cuda_points, sizeof(Point) * polygon.num_points);
  hipMemcpy(cuda_points, polygon.points, sizeof(Point) * polygon.num_points, hipMemcpyHostToDevice);

  return cuda_points;
}

Polygon* polygons_to_cuda(Encoding* img) {
  Polygon* cuda_polygons;
  hipMalloc(&cuda_polygons, sizeof(Polygon) * img->num_polygons);

  for (int i = 0; i < img->num_polygons; i++) {
    img->polygons[i].points = points_to_cuda(img->polygons[i]);
  }

  hipMemcpy(cuda_polygons, img->polygons, sizeof(Polygon) * img->num_polygons, hipMemcpyHostToDevice);

  return cuda_polygons;
}

Encoding* encoding_to_cuda(Encoding* img) {
  Encoding* cuda_img;
  hipMalloc(&cuda_img, sizeof(Encoding));

  img->polygons = polygons_to_cuda(img);
  hipMemcpy(cuda_img, img, sizeof(Encoding), hipMemcpyHostToDevice);

  return cuda_img;
}

extern "C" void cuda_render(Encoding img, Color* output, bool antialias) {
  uint N = img.width * img.height;
  size_t size = N * sizeof(Color);

  Color* cuda_output;
  hipMalloc(&cuda_output, size);
  hipMemcpy(cuda_output, output, size, hipMemcpyHostToDevice);

  Encoding* cuda_img = encoding_to_cuda(&img);

  dim3 threadsPerBlock(256, 1);
  dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x);
  render_kernel<<<blocksPerGrid, threadsPerBlock>>>(cuda_img, cuda_output, antialias);

  hipMemcpy(output, cuda_output, size, hipMemcpyDeviceToHost);

  hipFree(cuda_output);
  hipFree(cuda_img);
}