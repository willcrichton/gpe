#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREADS_PER_BLOCK 128

typedef unsigned int u32;
typedef unsigned char u8;

typedef struct {
  u8 r;
  u8 g;
  u8 b;
} Color;

typedef struct {
  float x;
  float y;
} Point;

typedef struct {
  Point* points;
  u32 num_points;
  u8 r; u8 g; u8 b; u8 a;
  Point center;
  float max_dist;
} Polygon;

typedef struct {
  Polygon* polygons;
  u32 num_polygons;
  u32 width;
  u32 height;
  Polygon* dev_ptr;
} Encoding;

__device__ __inline__ u8 add(u32 older, u32 newer, u32 alpha) {
  u32 addend = newer * alpha / 255;
  if (addend + older > 255) { /*printf("%d, %d, %d\n", older, newer, alpha);*/ return 255; }
  else { return addend + older; }
}

__device__ __inline__ void blend(Color* old_color, Color* new_color, u32 alpha) {
  old_color->r = add(old_color->r, new_color->r, alpha);
  old_color->g = add(old_color->g, new_color->g, alpha);
  old_color->b = add(old_color->b, new_color->b, alpha);
}

__device__ __inline__ Color polycolor(Polygon poly, Point pt) {
  Color color;
  float x = pt.x - poly.center.x;
  float y = pt.y - poly.center.y;
  float scale = 1.0 - (x * x + y * y) / poly.max_dist;

  /*if (scale < 0.0 || scale >= 1.0) {
    printf("scale %f, max_dist %f, point (%f, %f), center (%f, %f)\n", scale, poly.max_dist, pt.x, pt.y, poly.center.x, poly.center.y);
    }*/

  color.r = poly.r * scale;
  color.g = poly.g * scale;
  color.b = poly.b * scale;
  return color;
}

__device__ bool query(Point pt, Polygon poly, bool antialias) {
  bool inside = false;
  for (int i = 0; i < poly.num_points; i++) {
    Point a = poly.points[i], b = poly.points[(i + 1) % poly.num_points];
    if ((a.y > pt.y) != (b.y > pt.y) &&
        (pt.x < (b.x - a.x) * (pt.y - a.y) / (b.y - a.y) + a.x)) {
      inside = !inside;
    }
  }

  return inside;
}

__global__ void render_kernel(Encoding* img, Color* output, bool antialias) {
  int pixel = blockDim.x * blockIdx.x + threadIdx.x;
  if (pixel >= img->width * img->height) return;

  Point pt = { pixel % img->width, pixel / img->width };

  for (int i = 0; i < img->num_polygons; i++) {
    Polygon polygon = img->polygons[i];

    if (query(pt, polygon, antialias)) {
      blend(&output[pixel], &polycolor(polygon, pt), polygon.a);
    }
  }
}

Point* points_to_cuda(Polygon polygon) {
  Point* cuda_points;
  hipMalloc(&cuda_points, sizeof(Point) * polygon.num_points);
  hipMemcpy(cuda_points, polygon.points, sizeof(Point) * polygon.num_points, hipMemcpyHostToDevice);

  return cuda_points;
}

Polygon* polygons_to_cuda(Encoding* img) {
  Polygon* cuda_polygons;
  hipMalloc(&cuda_polygons, sizeof(Polygon) * img->num_polygons);

  for (int i = 0; i < img->num_polygons; i++) {
    float max_dist = 0.0;
    for (int j = 0; j < img->polygons[i].num_points; j++) {
      float x = img->polygons[i].points[j].x - img->polygons[i].center.x;
      float y = img->polygons[i].points[j].y - img->polygons[i].center.y;
      float dist = x * x + y * y;
      if (dist > max_dist) {
        max_dist = dist;
      }
    }

    img->polygons[i].max_dist = max_dist;
    img->polygons[i].points = points_to_cuda(img->polygons[i]);
  }

  hipMemcpy(cuda_polygons, img->polygons, sizeof(Polygon) * img->num_polygons, hipMemcpyHostToDevice);

  return cuda_polygons;
}

Encoding* encoding_to_cuda(Encoding* img) {
  Encoding* cuda_img;
  hipMalloc(&cuda_img, sizeof(Encoding));

  Polygon* tmp = img->polygons;
  img->polygons = polygons_to_cuda(img);
  hipMemcpy(cuda_img, img, sizeof(Encoding), hipMemcpyHostToDevice);

  img->dev_ptr = img->polygons;
  img->polygons = tmp;
  return cuda_img;
}

void encoding_free(Encoding* img) {
  for (int i = 0; i < img->num_polygons; i++) {
    hipFree(img->polygons[i].points);
  }

  hipFree(img->dev_ptr);
}

extern "C" void cuda_render(Encoding img, Color* output, bool antialias) {
  u32 N = img.width * img.height;
  size_t size = N * sizeof(Color);

  Color* cuda_output;
  hipMalloc(&cuda_output, size);
  hipMemcpy(cuda_output, output, size, hipMemcpyHostToDevice);

  Encoding* cuda_img = encoding_to_cuda(&img);

  dim3 threadsPerBlock(THREADS_PER_BLOCK, 1);
  dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x);
  render_kernel<<<blocksPerGrid, threadsPerBlock>>>(cuda_img, cuda_output, antialias);

  hipMemcpy(output, cuda_output, size, hipMemcpyDeviceToHost);

  encoding_free(&img);
  hipFree(cuda_img);
  hipFree(cuda_output);
}